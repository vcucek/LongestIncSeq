#include "hip/hip_runtime.h"

#include "SequenceAnalyser.hpp"
#include <iostream>

static int THREADS_PER_BLOCK = 256;

typedef struct
{
    Interval incr[3];
} BatchResult;

static __device__ int getGlobalComputeIndex()
{
    return blockIdx.x * blockDim.x + threadIdx.x;
}

static __global__ void findIntervalsKernel(
    const float *data,
    long dataSize,
    long batchSize,
    BatchResult *results)
{

    int computeIndex = getGlobalComputeIndex();

    // Clear result struct
    BatchResult *res = &results[computeIndex];
    res->incr[0].startIndex = -1;
    res->incr[0].endIndex = -1;
    res->incr[1].startIndex = -1;
    res->incr[1].endIndex = -1;
    res->incr[2].startIndex = -1;
    res->incr[2].endIndex = -1;

    int dataBatchStartIndex = computeIndex * batchSize;
    if (dataBatchStartIndex > dataSize - 1)
    {
        return;
    }
    int dataBatchEndIndex = dataBatchStartIndex + batchSize;
    if (dataBatchEndIndex > dataSize)
    {
        dataBatchEndIndex = dataSize;
    }

    // Clear result struct
    float last_value = data[dataBatchStartIndex];
    float value;

    // Interval potentially crossing batch start boundary
    res->incr[0].startIndex = dataBatchStartIndex;
    res->incr[2].startIndex = dataBatchEndIndex - 1;
    res->incr[2].endIndex = dataBatchEndIndex;
    Interval current = res->incr[0];
    bool has_current = true;

    for (int i = dataBatchStartIndex + 1; i < dataBatchEndIndex; i++)
    {
        value = data[i];
        if (value > last_value && !has_current)
        {
            current.startIndex = i - 1;
            current.endIndex = -1;
            has_current = true;
        }
        if (value <= last_value && has_current)
        {
            current.endIndex = i;
            // Reevaluate longest interval in this batch
            int longestLength = res->incr[1].endIndex - res->incr[1].startIndex;
            int currentLength = current.endIndex - current.startIndex;
            if (currentLength > longestLength)
            {
                res->incr[1] = current;
            }
            if (current.startIndex == dataBatchStartIndex)
            {
                res->incr[0] = current;
            }
            has_current = false;
        }
        last_value = value;
    }

    // Interval potentially crossing batch end boundary
    if (has_current)
    {
        current.endIndex = dataBatchEndIndex;

        res->incr[2].startIndex = current.startIndex;
        res->incr[2].endIndex = current.endIndex;

        // Reevaluate longest interval in this batch
        int longestLength = res->incr[1].endIndex - res->incr[1].startIndex;
        int currentLength = current.endIndex - current.startIndex;
        if (currentLength > longestLength)
        {
            res->incr[1] = current;
        }
        if (current.startIndex == dataBatchStartIndex)
        {
            res->incr[0] = current;
        }
    }
}

Interval
SequenceAnalyserCUDA::calcLongestIncInterval(FloatSequence &seq)
{

    // Check CUDA
    hipError_t error;
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr 
            << "ERROR: Initializing CUDA: " << hipGetErrorString(error) 
            << ", Exiting...(code 1)" << std::endl;
        exit(1);
    }

    // Allocate device input data
    float *devFloatArray = NULL;
    long devFloatArraySize = seq.getBufferSize() / sizeof(float);
    hipMalloc(&devFloatArray, seq.getBufferSize());
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr 
            << "ERROR: Allocating device input data: " << hipGetErrorString(error) 
            << ", Exiting...(code 1)" << std::endl;
        exit(1);
    }
    // Copy input data from host to device
    hipMemcpy(devFloatArray, seq.getBuffer(), seq.getBufferSize(), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr 
            << "ERROR: Copying input data from host to device: " << hipGetErrorString(error) 
            << ", Exiting...(code 1)" << std::endl;
        exit(1);
    }

    int tasks = (int)(seq.getSize() / batchSize) + 1;
    int blocks = (int)(tasks / THREADS_PER_BLOCK) + 1;

    // Allocate device output data
    int batchResultArraySize = tasks;
    BatchResult *batchResultArray = NULL;
    batchResultArray = (BatchResult *)calloc(batchResultArraySize, sizeof(BatchResult));
    BatchResult *devBatchResultArray = NULL;
    hipMalloc(&devBatchResultArray, batchResultArraySize * sizeof(BatchResult));
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr 
            << "ERROR: Allocating device output data memory block: " << hipGetErrorString(error) 
            << ", Exiting...(code 1)" << std::endl;
        exit(1);
    }

    // Run CUDA kernel
    findIntervalsKernel<<<blocks, THREADS_PER_BLOCK>>>(
        devFloatArray, devFloatArraySize,
        batchSize,
        devBatchResultArray);

    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr 
            << "ERROR: Running findIntervalsKernel: " << hipGetErrorString(error) 
            << ", Exiting...(code 1)" << std::endl;
        exit(1);
    }

    // Copy output data from device to host
    hipMemcpy(batchResultArray, devBatchResultArray,
               batchResultArraySize * sizeof(BatchResult),
               hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr 
            << "ERROR: Copying findIntervalsKernel results from device to host: " << hipGetErrorString(error) 
            << ", Exiting...(code 1)" << std::endl;
        exit(1);
    }

    // Filter longest interval from candidates
    Interval interCurrent = {0, 1};
    Interval interLongest = {0, 1};

    for (int i = 0; i < batchResultArraySize; i++)
    {
        for (int s = 0; s < 3; s++)
        {
            Interval inter = batchResultArray[i].incr[s];
            if (inter.startIndex >= 0 && inter.endIndex >= 1)
            {

                if (s == 0 && interCurrent.endIndex == inter.startIndex)
                {
                    float endValue = seq.getValue(interCurrent.endIndex - 1);
                    float startValue = seq.getValue(inter.startIndex);
                    if (startValue > endValue)
                    {
                        // Interval can be extended
                        interCurrent.endIndex = inter.endIndex;
                        if (interCurrent.endIndex - interCurrent.startIndex > interLongest.endIndex - interLongest.startIndex)
                        {
                            interLongest = interCurrent;
                        }
                    }
                }
                else if (inter.endIndex > interCurrent.endIndex)
                {
                    interCurrent = inter;
                    // Check and set if current interval is longer
                    if (interCurrent.endIndex - interCurrent.startIndex > interLongest.endIndex - interLongest.startIndex)
                    {
                        interLongest = interCurrent;
                    }
                }
            }
        }
    }

    // Dispose allocated device and host memory
    hipFree(devBatchResultArray);
    hipFree(devFloatArray);
    free(batchResultArray);

    return interLongest;
}